
#include <hip/hip_runtime.h>
#include <iostream>
#include <string.h>
#include <stdio.h>
#include <unistd.h>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <cstdlib>

using namespace std;


__global__ void Radix(char * d_in, char * d_out, int * d_scan,  int * d_scatter, int * d_past, int size)
{
    int idx = threadIdx.x ;
    int total;

    for(int j = 0; j<8; j++)
    {
        if (j==0)
        {
            //Creates a Inclusive scan of where there are 0's (if there is a 0, it is incremented)
            d_past[idx] = (d_in[idx] & (1 << j)) ? 0 : 1;
            d_scan[idx] = d_past[idx];
        }
        else
        {
            //Creates a Inclusive scan of where there are 0's (if there is a 0, it is incremented)
            d_past[idx] = (d_in[d_scatter[idx]] & (1 << j)) ? 0 : 1;
            d_scan[idx] = d_past[idx];
        }

        __syncthreads();
        
        //(Scan jumping operation)
        for(int jump = 1; jump < size; jump *= 2)
        {

            if(idx - jump >= 0)
                d_scan[idx] = d_scan[idx] + d_past[idx-jump];

            d_past[idx] = d_scan[idx];

            __syncthreads();
        }

        //From the inclusive scan, the total amount of 0's is equal to the last place in the scan.
        total = d_scan[size - 1];

        __syncthreads();

        //Converts to a exclusive scan, will change to a belloch scan that is more work efficient
        if (idx != 0)
            d_scan[idx] = d_past[idx-1];
        else
            d_scan[0] = 0;

        __syncthreads();


        if (j==0)
        {
            if ((d_in[idx] & (1 << j)) == false)
                d_scatter[d_scan[idx]] = idx;

            else
                d_scatter[(idx + total) - d_scan[idx]] = idx;
        }
        else
        {
            if ((d_in[d_scatter[idx]] & (1 << j)) == false)
                d_past[d_scan[idx]] = idx;

            else
                d_past[(idx + total) - d_scan[idx]] = idx;
        }

        __syncthreads();

        if (j != 0)
            d_scatter[idx] = d_scatter[d_past[idx]];

        __syncthreads();

    }

    d_out[idx] = d_in[d_scatter[idx]];
}

int main() {
    int size;

    //This grabs the file named 'test.txt' as input data for the BWT encode, if there is no filed named test.txt, it will throw an error message
    ifstream inputfile("1024.txt", ios::binary);

    if(!inputfile.is_open())
    {
        cout << "ERR: -- Incorrect file name\n";
        exit(1);
    }

    //This calculates how many charecters are in the test.txt file. (size is decremented to exclude the EOF charecter)
    inputfile.seekg(0, inputfile.end);
    size = inputfile.tellg();
    inputfile.seekg(0, inputfile.beg);
    size--;

    //A dynamic memory pointer will be created to put the .txt file into memory so we can manipulate the data.
    char* input = new char [size];

    //reads the data into the char array, if for any reason the chars cannoty be read, it will throw an error message

    inputfile.read(input,size);
    if (!inputfile)
    {
        cout << "ERR: -- Charecters have not been read successfully" << endl;
        exit(1);
    }
    inputfile.close();

    //Declare the amount of memory needed for both types of arrays
    const int Array_char = size * sizeof(char);
    const int Array_int = size * sizeof(int);

    // declare GPU memory pointers
    char * d_in;
    char * d_out;
    int * d_scan;
    int * d_scatter;
    int * d_past;

    //declare array to hold to sorted list
    char h_out[size];

    // allocate GPU memory
    hipMalloc((void**) &d_in, Array_char);
    hipMalloc((void**) &d_out, Array_char);
    hipMalloc((void**) &d_scan, Array_int);
    hipMalloc((void**) &d_scatter, Array_int);
    hipMalloc((void**) &d_past, Array_int);

    // transfer the array to the GPU
    hipMemcpy(d_in, input, Array_char, hipMemcpyHostToDevice);

    // launch the kernel
    Radix<<< 1, size >>>(d_in, d_out, d_scan, d_scatter, d_past, size);

    // copy back the GPU array to the CPU array
    hipMemcpy(h_out, d_out, Array_char, hipMemcpyDeviceToHost);

    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_scan);
    hipFree(d_scatter);
    hipFree(d_past);

    delete [] input;

    for(int i=0; i<size; i++)
        printf("%c", h_out[i]);
    printf("\n");


    return 0;
}
