
#include <hip/hip_runtime.h>
#include <iostream>
#include <string.h>
#include <stdio.h>
#include <unistd.h>
#include <fstream>
#include <sstream>
#include <algorithm>
#include <cstdlib>

using namespace std;

__global__ void Radix(char * d_in, char * d_out, int * d_scan,  int * d_scatter, int * d_past, int size)
{
    int idx = threadIdx.x ;
    int total;

    for(int j = 0; j<8; j++)
    {
        //Creates a Inclusive scan of where there are 0's (if there is a 0, it is incremented)
        d_past[idx] = (d_in[idx] & (1 << j)) ? 0 : 1;
        d_scan[idx] = d_past[idx];
        __syncthreads();

        for(int jump = 1; jump < size; jump *= 2)
        {
            if(idx - jump >= 0)
                d_scan[idx] = d_scan[idx] + d_past[idx-jump];
            d_past[idx] = d_scan[idx];
            __syncthreads();
        }
        //From the inclusive scan, the total amount of 0's is equal to the last place in the scan.
        total = d_scan[size - 1];
        __syncthreads();

        //Converts to a exclusive scan, will change to a belloch scan that is more work efficient
        if (idx != 0)
            d_scan[idx] = d_past[idx-1];
        else
            d_scan[0] = 0;
        __syncthreads();

        if ((d_in[idx] & (1 << j)) == false)
            d_scatter[d_scan[idx]] = idx;
        else
            d_scatter[(idx + total) - d_scan[idx]] = idx;
        __syncthreads();

        d_in[idx] = d_out[d_scatter[idx]];
        d_out[idx] = d_in[idx];
    }
}

void get_size(int& size){
    ifstream inputfile("test.txt", ios::binary);

    if(!inputfile.is_open())
    {
        cout << "ERR: -- Incorrect file name\n";
        exit(1);
    }

    inputfile.seekg(0, inputfile.end);
    size = inputfile.tellg();
    inputfile.seekg(0, inputfile.beg);
    size--;
    inputfile.close();
}

void get_char(int size, char* input){
    ifstream inputfile("test.txt", ios::binary);
    inputfile.read(input,size);
    if (!inputfile.is_open())
    {
        cout << "ERR: -- Charecters have not been read successfully\n";
        exit(1);
    }
    inputfile.close();
}

int main(){
    int size;
    get_size(size);

    char* input = new char [size];
    get_char(size, input);

    //Declare the amount of memory needed for both types of arrays
    const int Array_char = size * sizeof(char);
    const int Array_int = size * sizeof(int);

    // declare GPU memory pointers
    char * d_in;
    char * d_out;
    int * d_scan;
    int * d_scatter;
    int * d_past;

    //declare array to hold to sorted list
    char h_out[size];

    // allocate GPU memory
    hipMalloc((void**) &d_in, Array_char);
    hipMalloc((void**) &d_out, Array_char);
    hipMalloc((void**) &d_scan, Array_int);
    hipMalloc((void**) &d_scatter, Array_int);
    hipMalloc((void**) &d_past, Array_int);

    // transfer the array to the GPU
    hipMemcpy(d_in, input, Array_char, hipMemcpyHostToDevice);
    hipMemcpy(d_out, input, Array_char, hipMemcpyHostToDevice);

    // launch the kernel
    Radix<<< 1, size >>>(d_in, d_out, d_scan, d_scatter, d_past, size);

    // copy back the GPU array to the CPU array
    hipMemcpy(h_out, d_out, Array_char, hipMemcpyDeviceToHost);

    for(int i=0; i<size; i++)
        printf("%c", h_out[i]);
    printf("\n");


    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_scan);
    hipFree(d_scatter);
    hipFree(d_past);

    delete [] input;

}
